#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>

// kernel function to add elements of two arrays
__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {

    int N = 1 << 20;
    float *x, *y;

    // allocate unified memory (cpu & gpu)
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialise x and y arrays on host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // prefetch x and y arrays to the gpu
    hipMemPrefetchAsync(x, N * sizeof(float), 0, 0);
    hipMemPrefetchAsync(y, N * sizeof(float), 0, 0);

    // run kernel on 1 million elements on gpu
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // wait for gpu to finish before accessing host
    hipDeviceSynchronize();

    // error checking (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // free memory
    hipFree(x);
    hipFree(y);

    return 0;
}